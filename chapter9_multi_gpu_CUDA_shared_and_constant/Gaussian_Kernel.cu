#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <opencv2/opencv.hpp>

extern "C" void CUDA_Gaussian_Filter(uchar *pcuSrc, uchar *pcuDst,
	int w, int h, float *cuGkernel, int kernel_size);

__global__
void cuda_Filter_2D(uchar * pSrcImage, uchar *pDstImage,
	int SrcWidth, int SrcHeight, float *pKernel, int KWidth, int KHeight)
{
	// ��ϰ� ������ �ּҿ� ���� ���� pixel�� index�� ����Ѵ� 
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int index = y * SrcWidth + x;
	int mSize = KWidth / 2;

	float temp = 0.f;

	// Serial�� �����ϰ� ����ó�� ���� �� �� pixel ���
	if (x >= KWidth / 2 && y >= KHeight / 2
		&& x < SrcWidth - KWidth / 2 && y < SrcHeight - KHeight / 2)
	{
		for (int j = -mSize; j <= mSize; j++) {
			for (int i = -mSize; i <= mSize; i++) {

				// float ���·� ��� �� ����
				temp += (float)pSrcImage[index + i + j * SrcWidth]
					* pKernel[i + mSize + (j + mSize) * KHeight];
			}
		}
		// ���� dst �̹������� uchar�� ���·� ����
		pDstImage[index] = (uchar)temp;
	}
	else {
		pDstImage[index] = 0;		// kernel size �ٱ��� �ȼ��� ����ó�� ����
	}
}

__global__
void cuda_shared_Filter_2D(uchar * pSrcImage, uchar *pDstImage,
	int SrcWidth, int SrcHeight, float *pKernel, int KWidth, int KHeight)
{
	// ������ shared �޸� ����
	extern __shared__ float shared[];

	// ��ϰ� ������ �ּҿ� ���� ���� pixel�� index�� ����Ѵ�
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int index = y * SrcWidth + x;
	int mSize = KWidth / 2;

	if (tx < KWidth && ty < KHeight)
	{
		// shared �޸𸮿� size��ŭ Ŀ���� �����Ѵ�
		shared[ty * KWidth + tx] = pKernel[ty * KWidth + tx];
	}
	__syncthreads();

	float temp = 0.f;

	// Serial�� �����ϰ� ����ó�� ���� �� �� pixel ���
	if (x >= KWidth / 2 && y >= KHeight / 2
		&& x < SrcWidth - KWidth / 2 && y < SrcHeight - KHeight / 2)
	{

		for (int j = -mSize; j <= mSize; j++) {
			for (int i = -mSize; i <= mSize; i++) {

				// float ���·� ��� �� ����
				temp += (float)pSrcImage[index + i + j * SrcWidth]
					* shared[i + mSize + (j + mSize) * KHeight];
			}
		}
		// ���� dst �̹������� uchar�� ���·� ����
		pDstImage[index] = (uchar)temp;
	}
	else {
		pDstImage[index] = 0;		// kernel size �ٱ��� �ȼ��� ����ó�� ����
	}
}

// Ŀ�� �����ŭ �̸� constant �޸� ����
__constant__ float constKernel[5 * 5];

__global__
void cuda_constant_Filter_2D(uchar * pSrcImage, uchar *pDstImage,
	int SrcWidth, int SrcHeight, int KWidth, int KHeight)
{
	// ��ϰ� ������ �ּҿ� ���� ���� pixel�� index�� ����Ѵ�
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int index = y * SrcWidth + x;
	int mSize = KWidth / 2;

	float temp = 0.f;

	// Serial�� �����ϰ� ����ó�� ���� �� �� pixel ���
	if (x >= KWidth / 2 && y >= KHeight / 2
		&& x < SrcWidth - KWidth / 2 && y < SrcHeight - KHeight / 2)
	{

		for (int j = -mSize; j <= mSize; j++) {
			for (int i = -mSize; i <= mSize; i++) {
				// constant kernel�� ���Ͽ� ���
				temp += (float)pSrcImage[index + i + j * SrcWidth]
					* constKernel[i + mSize + (j + mSize) * KHeight];
			}
		}
		// ���� dst �̹������� uchar�� ���·� ����
		pDstImage[index] = (uchar)temp;
	}
	else {
		pDstImage[index] = 0;		// kernel size �ٱ��� �ȼ��� ����ó�� ����
	}
}

void CUDA_Gaussian_Filter(uchar *pcuSrc, uchar *pcuDst,
	int w, int h, float *cuGkernel, int kernel_size) {
	// 16 x 16 �������� ��ϰ� �� grid ������ ����
	dim3 grid = dim3(w / 16, h / 16);
	dim3 block = dim3(16, 16);

	// �� pixel �� CUDA Gaussain Filter ����

	int c = 1;			// 0 : global / 1 : shared / 2 : constant

	if (c == 0)
		cuda_Filter_2D << < grid, block >> > (pcuSrc, pcuDst, w, h, cuGkernel, kernel_size, kernel_size);
	else if (c == 1)
		// Ŀ�θ�ŭ�� size�� shared memory�� �������� �Ҵ��Ѵ�
		cuda_shared_Filter_2D << < grid, block, sizeof(float) * 5 * 5 >> > (pcuSrc, pcuDst, w, h, cuGkernel, kernel_size, kernel_size);
	else if (c == 2) {
		hipMemcpyToSymbol(HIP_SYMBOL(constKernel), cuGkernel, sizeof(float)*kernel_size*kernel_size);
		cuda_constant_Filter_2D << < grid, block >> > (pcuSrc, pcuDst, w, h, kernel_size, kernel_size);
	}

	// �޸� ��ũ�γ����� ����
	hipDeviceSynchronize();
}